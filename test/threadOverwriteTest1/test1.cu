#include <stdio.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#define THREADS_PER_BLOCK 4

__global__ void checkOverwrite(int *darr, int length)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < length)
    {
        int value = darr[idx % 8];
        darr[idx % 8] = idx;

        // confirm with threadOverwrite
        // if (value != idx) {
        //     printf("Thread %d overwrote the value with %d\n", idx, darr[idx]);
        // }
    }
}

int main()
{
    int length = 16;
    int threadsPerBlock = THREADS_PER_BLOCK;
    int* device_data;

    hipMalloc((void **)&device_data, sizeof(int) * length);
    int check[16];

//     for (int i = 0; i < length; i++)
//     {
//         check[i] = i;
//     }

    // cudaMemcpy(device_data, check, length * sizeof(int), cudaMemcpyHostToDevice);

    int numBlocks = (length + threadsPerBlock - 1) / threadsPerBlock;
    checkOverwrite<<<numBlocks, threadsPerBlock>>>(device_data, length);

    hipMemcpy(check, device_data, length * sizeof(int), hipMemcpyDeviceToHost);

    // for (int i = 0; i < length; i++){
    //     printf( "%d \n", check[i]);
    // }

    // free memory
    hipFree(device_data);

    return 0;
}
