#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

__global__ void checkOverwrite(int val, int length)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < length)
    {
        int value = val;
        val = idx;

        // confirm with threadOverwrite
        // if (value != idx) {
        //     printf("Thread %d overwrote the value with %d\n", idx, darr[idx]);
        // }
    }
}

int main()
{
    int length = 1;
    int threadsPerBlock = 3;
    int device_data;

    hipMalloc((void *)&device_data, sizeof(int));
    int check = 0;

    // hipMemcpy(device_data, check, length * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (length + threadsPerBlock - 1) / threadsPerBlock;
    checkOverwrite<<<numBlocks, threadsPerBlock>>>(device_data, length);

    hipMemcpy(check, device_data, sizeof(int), hipMemcpyDeviceToHost);

    // for (int i = 0; i < length; i++){
    //     printf( "%d \n", check[i]);
    // }

    // free memory
    hipFree(device_data);

    return 0;
}
