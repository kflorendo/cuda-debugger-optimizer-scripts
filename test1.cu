#include <stdio.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

__global__ void checkOverwrite(int *darr, int length)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < length)
    {
        int value = darr[idx];
        darr[idx] = idx;

    }
}

int main()
{
    int length = 16;
    int threadsPerBlock = 4;
    int* device_data;

    hipMalloc((void **)&device_data, sizeof(int) * length);
    int check[16];

    for (int i = 0; i < length; i++)
    {
        check[i] = i;
    }

    hipMemcpy(device_data, check, length * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (length + threadsPerBlock - 1) / threadsPerBlock;
    checkOverwrite<<<numBlocks, threadsPerBlock>>>(device_data, length);

    hipMemcpy(check, device_data, length * sizeof(int), hipMemcpyDeviceToHost);

    // free memory
    hipFree(device_data);

    return 0;
}
